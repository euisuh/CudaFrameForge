#include "venturi_processor.h"

void enhanceVideoFrame(cv::Mat &inputMat, cv::VideoWriter &videoStream) {
    int matType = inputMat.type();
    int matStep = inputMat.step;
    inputMat.convertTo(inputMat, CV_32FC3);
    Npp32f *gpuInputMat, *gpuProcessedResult;
    size_t stride;

    // Memory allocation on GPU and transfer of input matrix
    hipMallocPitch(&gpuInputMat, &stride, 3 * inputMat.cols * sizeof(Npp32f), inputMat.rows);
    hipMemcpy2D(gpuInputMat, stride, inputMat.ptr(), inputMat.step, 3 * inputMat.cols * sizeof(Npp32f), inputMat.rows, hipMemcpyHostToDevice);

    // Set up buffers for channel separation
    Npp32f* gpuGreenFilterOutput;
    size_t filterStride;
    hipMallocPitch(&gpuProcessedResult, &filterStride, inputMat.cols * sizeof(Npp32f), inputMat.rows);
    hipMallocPitch(&gpuGreenFilterOutput, &filterStride, inputMat.cols * sizeof(Npp32f), inputMat.rows);

    NppiSize roi;
    roi.width = inputMat.cols;
    roi.height = inputMat.rows;

    // Channel isolation (green)
    nppiCopy_32f_C3C1R(gpuInputMat + 1, stride, gpuGreenFilterOutput, filterStride, roi);

    // Apply Scharr horizontal filter for edge detection
    nppiFilterScharrHoriz_32f_C1R(gpuGreenFilterOutput, filterStride, gpuProcessedResult, filterStride, roi);

    // Prepare the matrix for final output
    cv::Mat channelMat(inputMat.rows, inputMat.cols, CV_32FC1);
    cv::Mat grayMat(inputMat.rows, inputMat.cols, matType);

    hipMemcpy2D(channelMat.ptr(), channelMat.step, gpuProcessedResult, filterStride, inputMat.cols * sizeof(Npp32f), inputMat.rows, hipMemcpyDeviceToHost);
    channelMat.convertTo(channelMat, CV_8UC1);
    cv::cvtColor(channelMat, grayMat, cv::COLOR_GRAY2BGR);

    // Output the frame into the video stream
    videoStream.write(grayMat);

    // Release GPU resources
    hipFree(gpuInputMat);
    hipFree(gpuProcessedResult);
}

__host__ int main(int argc, char** argv) {
    std::cout << "Initializing Video Processing by Venturi Systems\n";

    if (argc != 3) {
        std::cerr << "Error: Expected two arguments, received " << argc - 1 << ". Please provide a source and destination video path.\n";
        return EXIT_FAILURE;
    }

    cv::VideoCapture captureDevice(argv[1]);
    if (!captureDevice.isOpened()) {
        std::cerr << "Failed to open the source video file.\n";
        return EXIT_FAILURE;
    }

    int captureWidth = static_cast<int>(captureDevice.get(cv::CAP_PROP_FRAME_WIDTH));
    int captureHeight = static_cast<int>(captureDevice.get(cv::CAP_PROP_FRAME_HEIGHT));
    double frameRate = captureDevice.get(cv::CAP_PROP_FPS);

    cv::VideoWriter outputStream(argv[2], cv::VideoWriter::fourcc('M', 'P', '4', 'V'), frameRate, cv::Size(captureWidth, captureHeight));
    if (!outputStream.isOpened()) {
        std::cerr << "Unable to create the output video file with the specified codec.\n";
        return EXIT_FAILURE;
    }

    cv::Mat frameBuffer;
    int frameCount = 0;
    while (captureDevice.read(frameBuffer)) {
        std::cout << "Enhancing Frame: " << frameCount << "\n";
        frameCount++;

        enhanceVideoFrame(frameBuffer, outputStream);
    }

    return EXIT_SUCCESS;
}
